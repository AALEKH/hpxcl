#include "hip/hip_runtime.h"
// Copyright (c)       2015 Patrick Diehl
//
// Distributed under the Boost Software License, Version 1.0. (See accompanying
// file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)


#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#include "opencl/benchmark_vector/timer.hpp"

#include "config.hpp"
#include "utils.hpp"

//###########################################################################
//Kernels
//###########################################################################

template<typename T>
__global__ void stencil(size_t count, T* in, T* out, T* s) {
	for (int i = (blockDim.x * blockIdx.x +1) + threadIdx.x; i < count - 1;
			i += gridDim.x * blockDim.x) {

		out[i] = s[0] * in[i-1] + s[1] * in[i] + s[2] * in[i+1];
	}
}

//###########################################################################
//Main
//###########################################################################

int main(int argc, char*argv[]) {

	if(argc != 2)
	{
		std::cout << "Usage: " << argv[0] << " #elements" << std::endl;
		exit(1);
	}

	double data = 0;
	size_t count = atoi(argv[1]);

	std::cout << count << " ";

	//Pointer
	TYPE* out;
	TYPE* out_dev;
	TYPE* in;
	TYPE* in_dev;
	TYPE* s;
	TYPE* s_dev;

	/*
	 * Malloc data on host and device
	 */
	timer_start();
	//Malloc Host
	hipHostMalloc((void**) &out, count * sizeof(TYPE));
	hipHostMalloc((void**) &in, count * sizeof(TYPE));
	hipHostMalloc((void**) &s, 3 * sizeof(TYPE));
	//Malloc Device
	hipMalloc((void**) &out_dev, count * sizeof(TYPE));
	hipMalloc((void**) &in_dev, count * sizeof(TYPE));
	hipMalloc((void**) &s_dev, 3 * sizeof(TYPE));

	//Initialize the data
	fillRandomVector(in, count);
	s[0] = 0.5;
	s[1] = 1.;
	s[2] = 0.5;

	/*
	 * Copy data
	 */

	hipMemcpy(in_dev, in, count * sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemcpy(s_dev, s, 3 * sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemcpy(out_dev, in, count * sizeof(TYPE), hipMemcpyHostToDevice);

	int gridsize = 1;
	int blocksize = 32;

	/*
	 * Kernel launch
	 */
	stencil<TYPE><<<gridsize, blocksize>>>(count, in_dev, out_dev, s_dev);
	hipDeviceSynchronize();

	/*
	 * Copy result back
	 */
	hipMemcpy(out, out_dev, count * sizeof(TYPE), hipMemcpyDeviceToHost);

	data += timer_stop();

	//Check the result
	std::cout << checkStencil(in,out,s, count) << " ";

	timer_start();
	/*
	 * Free
	 */
	hipHostFree(in);
	hipHostFree(s);
	hipHostFree(out);
	hipFree(out_dev);
	hipFree(in_dev);
	hipFree(s_dev);

	std:: cout << data + timer_stop() << std::endl;

	return EXIT_SUCCESS;
}
